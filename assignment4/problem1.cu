// nvcc -ccbin /usr/bin/g++-10 -lineinfo -res-usage -arch=sm_80 -std=c++11 problem1.cu -o problem1

#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

const uint64_t N = (64);
#define THRESHOLD (0.000001)
const uint64_t MAX_VAL = 1e6;

using std::cerr;
using std::cout;
using std::endl;

#define cudaCheckError(ans)                                                                        \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

__global__ void kernel1(const double *in, double *out) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  if (i > 0 && i < N - 1 && j > 0 && j < N - 1 && k > 0 && k < N - 1) {
    out[i * N * N + j * N + k] = 0.8 * (in[(i - 1) * N * N + j * N + k]
                                      + in[(i + 1) * N * N + j * N + k]
                                      + in[i * N * N + (j - 1) * N + k]
                                      + in[i * N * N + (j + 1) * N + k]
                                      + in[i * N * N + j * N + (k - 1)]
                                      + in[i * N * N + j * N + (k + 1)]);
  }
}

// TODO: Edit the function definition as required
__global__ void kernel2() {}

__host__ void stencil(const double *in, double *out) {
  for (int i = 1; i < N - 1; i++) {
    for (int j = 1; j < N - 1; j++) {
      for (int k = 1; k < N - 1; k++) {
        out[i * N * N + j * N + k] = 0.8 * (in[(i - 1) * N * N + j * N + k]
                                          + in[(i + 1) * N * N + j * N + k]
                                          + in[i * N * N + (j - 1) * N + k]
                                          + in[i * N * N + (j + 1) * N + k]
                                          + in[i * N * N + j * N + (k - 1)]
                                          + in[i * N * N + j * N + (k + 1)]);
      }
    }
  }
}

__host__ void check_result(const double* w_ref, const double* w_opt, const uint64_t size) {
  double maxdiff = 0.0, this_diff = 0.0;
  int numdiffs = 0;

  for (uint64_t i = 0; i < size; i++) {
    for (uint64_t j = 0; j < size; j++) {
      for (uint64_t k = 0; k < size; k++) {
        this_diff = w_ref[i + N * j + N * N * k] - w_opt[i + N * j + N * N * k];
        if (std::fabs(this_diff) > THRESHOLD) {
          numdiffs++;
          if (this_diff > maxdiff) {
            maxdiff = this_diff;
          }
        }
      }
    }
  }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD << "; Max Diff = " << maxdiff
         << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

void print_mat(double* A) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      for (int k = 0; k < N; ++k) {
        printf("%lf,", A[i * N * N + j * N + k]);
      }
      printf("      ");
    }
    printf("\n");
  }
}

double rtclock() { // Seconds
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main() {
  uint64_t SIZE = N * N * N;

  double *h_in = static_cast<double *>(malloc(SIZE * sizeof(double)));
  double *h_out_serial = static_cast<double *>(malloc(SIZE * sizeof(double)));
  double *h_out = static_cast<double *>(malloc(SIZE * sizeof(double)));

  for (int i = 0; i < SIZE; i++) {
    h_in[i] = rand() % MAX_VAL;
    h_out_serial[i] = 0.0;
    h_out[i] = 0.0;
  }

  double clkbegin = rtclock();
  stencil(h_in, h_out_serial);
  double clkend = rtclock();
  double cpu_time = clkend - clkbegin;
  cout << "Stencil time on CPU: " << cpu_time * 1000 << " msec" << endl;

  double *d_in;
  cudaCheckError(hipMalloc(&d_in, SIZE * sizeof(double)));
  double *d_out;
  cudaCheckError(hipMalloc(&d_out, SIZE * sizeof(double)));

  cudaCheckError(hipMemcpy(d_in, h_in, SIZE * sizeof(double),
                            hipMemcpyHostToDevice));

  hipEvent_t start, end;
  cudaCheckError(hipEventCreate(&start));
  cudaCheckError(hipEventCreate(&end));
  cudaCheckError(hipEventRecord(start));

  dim3 threadsPerBlock(1, 32, 32);
  dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y, N / threadsPerBlock.z);
  kernel1<<<numBlocks, threadsPerBlock>>>(d_in, d_out);
  
  cudaCheckError(hipEventRecord(end));
  cudaCheckError(hipEventSynchronize(end));

  cudaCheckError(hipMemcpy(h_out, d_out, SIZE * sizeof(double),
                            hipMemcpyDeviceToHost));

  check_result(h_out_serial, h_out, N);

  float kernel_time;
  cudaCheckError(hipEventElapsedTime(&kernel_time, start, end));
  std::cout << "Kernel 1 time (ms): " << kernel_time << "\n";

  // TODO: Fill in kernel2
  // TODO: Adapt check_result() and invoke
  cudaCheckError(hipEventElapsedTime(&kernel_time, start, end));
  std::cout << "Kernel 2 time (ms): " << kernel_time << "\n";

  // TODO: Free memory

  return EXIT_SUCCESS;
}
